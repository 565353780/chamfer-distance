#include "hip/hip_runtime.h"
#include "cukd_searcher.h"

#include <ATen/cuda/HIPContext.h>
#include <omp.h>

#define CUKD_BUILDER_INPLACE
#include <cukd/builder.h>
#include <cukd/fcp.h>

#include <tiny-cuda-nn/common_host.h>
#include <tiny-cuda-nn/gpu_memory.h>
#include <tiny-cuda-nn/multi_stream.h>

template <typename PointT>
struct OrderedPoint_traits : public cukd::default_data_traits<PointT> {
  using data_t = OrderedPoint<PointT>;
  using point_traits = cukd::point_traits<PointT>;
  using scalar_t = typename point_traits::scalar_t;

  static inline __device__ __host__ const PointT &
  get_point(const data_t &data) {
    return data.position;
  }

  static inline __device__ __host__ scalar_t get_coord(const data_t &data,
                                                       int dim) {
    return cukd::get_coord(get_point(data), dim);
  }

  enum { has_explicit_dim = false };
  static inline __device__ int get_dim(const data_t &) { return -1; }
};

// 复制内核
template <typename PointT>
__global__ void CopyKernel(OrderedPoint<PointT> *points, PointT *positions,
                           int n_batches, int n_points) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int nid = threadIdx.y + blockIdx.y * blockDim.y;

  if ((bid >= n_batches) || (nid >= n_points))
    return;

  // Row major
  int pid = bid * n_points + nid;
  points[pid].position = positions[pid];
  // Batch local index
  points[pid].idx = nid;
}

// 最近点查询内核
template <typename FloatT, typename PointT>
__global__ void
ClosestPointKernel(FloatT *d_dists, int *d_indices, PointT *d_queries,
                   int n_batches, int n_queries,
                   const cukd::box_t<PointT> *d_bounds,
                   OrderedPoint<PointT> *d_nodes, int n_points) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int mid = threadIdx.y + blockIdx.y * blockDim.y;

  if ((bid >= n_batches) || (mid >= n_queries))
    return;

  // Row major
  int qid = bid * n_queries + mid;
  PointT queryPos = d_queries[qid];
  cukd::FcpSearchParams params;
  // Local closest index
  int closestID =
      cukd::cct::fcp<OrderedPoint<PointT>, OrderedPoint_traits<PointT>>(
          queryPos, *(d_bounds + bid), d_nodes + bid * n_points, n_points,
          params);
  int pid = bid * n_points + closestID;
  int idx = d_nodes[pid].idx;
  PointT inputPos = d_nodes[pid].position;

  float x_diff = queryPos.x - inputPos.x;
  float y_diff = queryPos.y - inputPos.y;
  float z_diff = queryPos.z - inputPos.z;

  d_dists[qid] = x_diff * x_diff + y_diff * y_diff + z_diff * z_diff;
  d_indices[qid] = idx;
}

// CUDA函数：分配内存并构建KD树
template <typename FloatT, typename PointT, uint32_t THREAD_POOL,
          uint32_t BATCH_SIZE_B, uint32_t BATCH_SIZE_N, uint32_t BATCH_SIZE_M>
void buildKDTree(const torch::Tensor &input, void **d_nodes, void **d_bounds) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  uint32_t numBatches = input.size(0);
  uint32_t numInput = input.size(1);

  // We must copy because implicit tree will re-arange input data
  CUKD_CUDA_CHECK(hipMallocAsync(
      d_nodes, numBatches * numInput * sizeof(OrderedPoint<PointT>), stream));

  // **IMPORTANT** We cannot loop, as data is in device memory
  CopyKernel<<<dim3(cukd::divRoundUp(numBatches, BATCH_SIZE_B),
                    cukd::divRoundUp(numInput, BATCH_SIZE_N)),
               dim3(BATCH_SIZE_B, BATCH_SIZE_N), 0, stream>>>(
      static_cast<OrderedPoint<PointT> *>(d_nodes),
      reinterpret_cast<PointT *>(input.data_ptr<FloatT>()), numBatches,
      numInput);
  hipStreamSynchronize(stream);

  CUKD_CUDA_CHECK(hipMallocAsync(
      d_bounds, numBatches * sizeof(cukd::box_t<PointT>), stream));

  // Build tree in parallel
  tcnn::SyncedMultiStream syncedStreams(stream, THREAD_POOL);
  omp_set_num_threads(THREAD_POOL);
#pragma omp parallel for schedule(dynamic)
  for (int bid = 0; bid < numBatches; bid++) {
    int tid = omp_get_thread_num();
    cukd::buildTree<OrderedPoint<PointT>, OrderedPoint_traits<PointT>>(
        static_cast<OrderedPoint<PointT> *>(*d_nodes) + bid * numInput,
        numInput, static_cast<cukd::box_t<PointT> *>(*d_bounds) + bid,
        syncedStreams.get(tid));
    hipStreamSynchronize(syncedStreams.get(tid));
  }
}

// CUDA函数：查询KD树
template <typename FloatT, typename PointT, uint32_t THREAD_POOL,
          uint32_t BATCH_SIZE_B, uint32_t BATCH_SIZE_N, uint32_t BATCH_SIZE_M>
std::vector<torch::Tensor> queryKDTree(void *d_nodes, void *d_bounds,
                                       const torch::Tensor &query,
                                       const uint32_t &n_points) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  uint32_t numBatches = query.size(0);
  uint32_t numQueries = query.size(0);

  const torch::TensorOptions distOpts =
      torch::TensorOptions().dtype(query.dtype()).device(query.device());
  torch::Tensor dists = torch::zeros({numBatches, numQueries}, distOpts);

  const torch::TensorOptions idxOpts =
      torch::TensorOptions().dtype(torch::kInt32).device(query.device());
  torch::Tensor idxs = torch::zeros({numBatches, numQueries}, idxOpts);

  ClosestPointKernel<<<dim3(cukd::divRoundUp(numBatches, BATCH_SIZE_B),
                            cukd::divRoundUp(numQueries, BATCH_SIZE_M)),
                       dim3(BATCH_SIZE_B, BATCH_SIZE_M), 0, stream>>>(
      dists.data_ptr<FloatT>(), idxs.data_ptr<int>(),
      reinterpret_cast<PointT *>(query.data_ptr<FloatT>()), numBatches,
      numQueries, static_cast<cukd::box_t<PointT> *>(d_bounds),
      static_cast<OrderedPoint<PointT> *>(d_nodes), n_points);
  hipStreamSynchronize(stream);

  return {dists, idxs};
}
