#include "hip/hip_runtime.h"
#include "cukd_searcher.h"
#include <ATen/cuda/HIPContext.h>
#include <cukd/builder.h>
#include <cukd/fcp.h>

// 定义OrderedPoint_traits
template <typename T>
struct OrderedPoint_traits : public cukd::default_data_traits<T> {
  using data_t = OrderedPoint<T>;
  using point_traits = cukd::point_traits<T>;
  using scalar_t = typename point_traits::scalar_t;

  static inline __device__ __host__ const T &get_point(const data_t &data) {
    return data.position;
  }

  static inline __device__ __host__ scalar_t get_coord(const data_t &data,
                                                       int dim) {
    return cukd::get_coord(get_point(data), dim);
  }

  enum { has_explicit_dim = false };
  static inline __device__ int get_dim(const data_t &) { return -1; }
};

// 复制内核
template <typename T>
__global__ void CopyKernel(OrderedPoint<T> *points, T *positions,
                           int n_points) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n_points)
    return;
  points[tid].position = positions[tid];
  points[tid].idx = tid;
}

// 最近点查询内核
template <typename FloatT, typename T>
__global__ void ClosestPointKernel(FloatT *d_dists, int *d_indices,
                                   T *d_queries, int numQueries,
                                   const cukd::box_t<T> *d_bounds,
                                   OrderedPoint<T> *d_nodes, int numNodes) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numQueries)
    return;

  T queryPos = d_queries[tid];
  cukd::FcpSearchParams params;

  int closestID = cukd::cct::fcp<OrderedPoint<T>, OrderedPoint_traits<T>>(
      queryPos, *d_bounds, d_nodes, numNodes, params);

  int idx = d_nodes[closestID].idx;
  T inputPos = d_nodes[closestID].position;

  float x_diff = queryPos.x - inputPos.x;
  float y_diff = queryPos.y - inputPos.y;
  float z_diff = queryPos.z - inputPos.z;
  d_dists[tid] = x_diff * x_diff + y_diff * y_diff + z_diff * z_diff;

  d_indices[tid] = idx;
}

// CUDA函数：分配内存并构建KD树
void *CUKDSearcher::allocateAndBuildKDTree(const torch::Tensor &points,
                                           void **d_bounds) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  constexpr uint32_t BATCH_SIZE = 128;

  // 获取点云数量
  uint32_t numInput = points.size(0);

  // 分配内存
  void *d_input = nullptr;
  hipMallocAsync(&d_input, numInput * sizeof(OrderedPoint<PointT>), stream);
  hipMallocAsync(d_bounds, sizeof(cukd::box_t<PointT>), stream);

  // 复制数据
  CopyKernel<<<cukd::divRoundUp(numInput, BATCH_SIZE), BATCH_SIZE, 0, stream>>>(
      static_cast<OrderedPoint<PointT> *>(d_input),
      reinterpret_cast<PointT *>(points.data_ptr<float>()), numInput);

  // 构建KD树
  cukd::buildTree<OrderedPoint<PointT>, OrderedPoint_traits<PointT>>(
      static_cast<OrderedPoint<PointT> *>(d_input), numInput,
      static_cast<cukd::box_t<PointT> *>(*d_bounds), stream);

  // 同步确保构建完成
  hipStreamSynchronize(stream);

  return d_input;
}

// CUDA函数：查询KD树
std::vector<torch::Tensor>
CUKDSearcher::queryKDTree(void *d_input, void *d_bounds,
                          const torch::Tensor &points, int numInput) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  using FloatT = float;
  constexpr uint32_t BATCH_SIZE = 128;

  // 获取查询点数量
  uint32_t numQueries = points.size(0);

  // 创建输出张量
  const torch::TensorOptions distOpts =
      torch::TensorOptions().dtype(points.dtype()).device(points.device());
  torch::Tensor dists = torch::zeros({numQueries}, distOpts);

  const torch::TensorOptions idxOpts =
      torch::TensorOptions().dtype(torch::kInt32).device(points.device());
  torch::Tensor idxs = torch::zeros({numQueries}, idxOpts);

  // 执行查询
  ClosestPointKernel<<<cukd::divRoundUp(numQueries, BATCH_SIZE), BATCH_SIZE, 0,
                       stream>>>(
      dists.data_ptr<FloatT>(), idxs.data_ptr<int>(),
      reinterpret_cast<PointT *>(points.data_ptr<FloatT>()), numQueries,
      static_cast<cukd::box_t<PointT> *>(d_bounds),
      static_cast<OrderedPoint<PointT> *>(d_input), numInput);

  // 同步确保查询完成
  hipStreamSynchronize(stream);

  return {dists, idxs};
}
